
#include <hip/hip_runtime.h>
#define NUMTHREADS 16
#define THREADWORK 32

extern "C"
__global__ void gpuKendall(const float * a, size_t na, 
	const float * b, size_t nb, size_t sampleSize, double * results) 
{
	size_t 
		i, j, tests, 
		tx = threadIdx.x, ty = threadIdx.y, 
		bx = blockIdx.x, by = blockIdx.y,
		rowa = bx * sampleSize, rowb = by * sampleSize;
	float 
		discordant, concordant = 0.f, 
		numer, denom;

	__shared__ float threadSums[NUMTHREADS*NUMTHREADS];

	for(i = tx; i < sampleSize; i += NUMTHREADS) {
		for(j = i+1+ty; j < sampleSize; j += NUMTHREADS) {
			tests = ((a[rowa+j] >  a[rowa+i]) && (b[rowb+j] >  b[rowb+i]))
				+ ((a[rowa+j] <  a[rowa+i]) && (b[rowb+j] <  b[rowb+i])) 
				+ ((a[rowa+j] == a[rowa+i]) && (b[rowb+j] == b[rowb+i])); 
			concordant = concordant + (float)tests;
		}
	}
	threadSums[tx*NUMTHREADS+ty] = concordant;

	__syncthreads();
	for(i = NUMTHREADS >> 1; i > 0; i >>= 1) {
		if(ty < i) 
			threadSums[tx*NUMTHREADS+ty] += threadSums[tx*NUMTHREADS+ty+i];
		__syncthreads();
	}
  for(i = NUMTHREADS >> 1; i > 0; i >>= 1) {
    if((tx < i) && (ty == 0))
      threadSums[tx*NUMTHREADS] += threadSums[(tx+i)*NUMTHREADS];
    __syncthreads();
  }

	if((tx == 0) && (ty == 0)) {
		concordant = threadSums[0];
		denom = (float)sampleSize;
		denom = (denom * (denom - 1.f)) / 2.f; discordant = denom - concordant;
		numer = concordant - discordant;
		results[by*na+bx] = ((double)numer)/((double)denom);
	}
}
