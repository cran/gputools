#include "cuseful.h"
#include "R.h"
#include "kendall.h"
#include "hip/hiprtc.h"
#include "hip/hip_runtime.h"

#define NUMTHREADS 16

#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      error("\nerror: %d failed with error %s\n", x,              \
            hiprtcGetErrorString(result));                         \
    }                                                             \
  } while(0)

#define CUDA_SAFE_CALL(x)                                         \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      error("\nerror: %d failed with error %s\n", x, msg);        \
    }                                                             \
  } while(0)

void masterKendall(const float * x,  size_t nx, 
  const float * y, size_t ny,
  size_t sampleSize, double * results,
  const char * kernel_src)
{
	size_t 
		outputLength = nx * ny, outputBytes = outputLength*sizeof(double),
		xBytes = nx*sampleSize*sizeof(float), 
		yBytes = ny*sampleSize*sizeof(float); 
	float
		* gpux, * gpuy; 
	double
		* gpuResults;
	dim3
		initGrid(nx, ny), initBlock(NUMTHREADS, NUMTHREADS);

	hipMalloc((void **)&gpux, xBytes);
	hipMalloc((void **)&gpuy, yBytes);
	checkCudaError("input vector space allocation");

	hipMemcpy(gpux, x, xBytes, hipMemcpyHostToDevice);
	hipMemcpy(gpuy, y, yBytes, hipMemcpyHostToDevice);
	checkCudaError("copying input vectors to gpu");

	hipMalloc((void **)&gpuResults, outputBytes);
	checkCudaError("allocation of space for result matrix");

  hiprtcProgram prog;

  NVRTC_SAFE_CALL(
      hiprtcCreateProgram(&prog,  // prog
        kernel_src,              // buffer
        "kendall",               // name
        0,                       // numHeaders
        NULL,                    // headers
        NULL));                  // includeNames

  hiprtcResult compileResult = hiprtcCompileProgram(prog, 0, NULL);
  if (compileResult != HIPRTC_SUCCESS) error("cuda kernel compile failed");

  //  Obtain PTX from the program.
  size_t ptxSize;
  NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));

  char *ptx = new char[ptxSize];
  NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));

  //  Destroy the program.
  NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

  //  Load the generated PTX and get a handle to the SAXPY kernel.
//  hipDevice_t cuDevice;
//  hipCtx_t context;

  CUDA_SAFE_CALL(hipInit(0));

//  CUDA_SAFE_CALL(hipDeviceGet(&cuDevice, 0));
//  CUDA_SAFE_CALL(hipCtxCreate(&context, 0, cuDevice));

  hipModule_t module;
  CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));

  hipFunction_t kernel;
  CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, "gpuKendall"));

  // execute kendall kernel
  void *args[] =
    { &gpux
    , &nx
    , &gpuy
    , &ny
    , &sampleSize
    , &gpuResults
    };

  CUDA_SAFE_CALL(
    hipModuleLaunchKernel(kernel,
      nx, ny, 1,                  // grid dim
      NUMTHREADS, NUMTHREADS, 1,  // block dim
      0, NULL,                    // shared mem and stream
      args, 0));                  // arguments
  CUDA_SAFE_CALL(hipCtxSynchronize());

  hipFree(gpux);
  hipFree(gpuy);
  hipMemcpy(results, gpuResults, outputBytes, hipMemcpyDeviceToHost);
  hipFree(gpuResults);
  checkCudaError("copying results from gpu and cleaning up");

  CUDA_SAFE_CALL(hipModuleUnload(module));
//  CUDA_SAFE_CALL(hipCtxDestroy(context));
}
