#include "hip/hip_runtime.h"
#include<Rmath.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<cuseful.h>
#include<granger.h>

#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM	16

#define FALSE 0
#define TRUE !FALSE

// mX has order rows x cols
// vectY has length rows
__global__ void getRestricted(int countx, int county, int rows, int cols, 
	float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim,
	float * mR, int mRdim, float * vectB, int vectBdim) {

	int 
		m = blockIdx.x * THREADSPERDIM + threadIdx.x, n,
		i, j, k;
	float 
		sum, invnorm,
		* X, * Y, * Q, * R, * B,
		* coli, * colj, 
		* colQ, * colX;

	if(m >= county) return;
	if(m == 1) n = 0;
	else n = 1;

	X = mX + (m * mXdim);
	// initialize the intercepts
	for(i = 0; i < rows; i++)
		X[i] = 1.f;

	Y = vY + (m * countx + n) * vYdim;
	B = vectB + m * vectBdim;
	Q = mQ + m * mQdim;
	R = mR + m * mRdim;

	// initialize Q with X ...
	for(i = 0; i < rows; i++) {
		for(j = 0; j < cols; j++)
			Q[i+j*rows] = X[i+j*rows];
	}

	// gramm-schmidt process to find Q
	for(j = 0; j < cols; j++) {
		colj = Q+rows*j;
		for(i = 0; i < j; i++) {
			coli = Q+rows*i;
			sum = 0.f;
			for(k = 0; k < rows; k++)
				sum += coli[k] * colj[k];
			for(k = 0; k < rows; k++)
				colj[k] -= sum * coli[k];
		}
		sum = 0.f;
		for(i = 0; i < rows; i++)
			sum += colj[i] * colj[i];
		invnorm = 1.f / sqrtf(sum);
		for(i = 0; i < rows; i++)
			colj[i] *= invnorm;
	}
	for(i = cols-1; i > -1; i--) {
		colQ = Q+i*rows;
		// matmult Q * X -> R
		for(j = 0; j < cols; j++) {
			colX = X+j*rows;
			sum = 0.f;
			for(k = 0; k < rows; k++)
				sum += colQ[k] * colX[k];
			R[i+j*cols] = sum;
		}
		sum = 0.f;
		// compute the vector Q^t * Y -> B
		for(j = 0; j < rows; j++) 
			sum += colQ[j] * Y[j];
		// back substitution to find the x for Rx = B
		for(j = cols-1; j > i; j--)
			sum -= R[i+j*cols] * B[j];

		B[i] = sum / R[i+i*cols];
	}
}

// mX has order rows x cols
// vectY has length rows
__global__ void getUnrestricted(int countx, int county, int rows, int cols, 
	float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim,
	float * mR, int mRdim, float * vectB, int vectBdim) {

	int 
		n = blockIdx.x * THREADSPERDIM + threadIdx.x, 
		m = blockIdx.y * THREADSPERDIM + threadIdx.y, 
		i, j, k;
	float 
		sum, invnorm,
		* X, * Y, * Q, * R, * B,
		* coli, * colj, 
		* colQ, * colX;
	if((m >= county) || (n >= countx)) return;

	X = mX + (m * countx + n) * mXdim;
	// initialize the intercepts
	for(i = 0; i < rows; i++) 
		X[i] = 1.f;

	Y = vY + (m*countx+n) * vYdim;
	B = vectB + (m*countx+n) * vectBdim;
	Q = mQ + (m*countx+n) * mQdim;
	R = mR + (m*countx+n) * mRdim;

	// initialize Q with X ...
	for(i = 0; i < rows; i++) {
		for(j = 0; j < cols; j++)
			Q[i+j*rows] = X[i+j*rows];
	}

	// gramm-schmidt process to find Q
	for(j = 0; j < cols; j++) {
		colj = Q+rows*j;
		for(i = 0; i < j; i++) {
			coli = Q+rows*i;
			sum = 0.f;
			for(k = 0; k < rows; k++)
				sum += coli[k] * colj[k];
			for(k = 0; k < rows; k++)
				colj[k] -= sum * coli[k];
		}
		sum = 0.f;
		for(i = 0; i < rows; i++)
			sum += colj[i] * colj[i];
		invnorm = 1.f / sqrtf(sum);
		for(i = 0; i < rows; i++)
			colj[i] *= invnorm;
	}
	for(i = cols-1; i > -1; i--) {
		colQ = Q+i*rows;
		// matmult Q * X -> R
		for(j = 0; j < cols; j++) {
			colX = X+j*rows;
			sum = 0.f;
			for(k = 0; k < rows; k++)
				sum += colQ[k] * colX[k];
			R[i+j*cols] = sum;
		}
		sum = 0.f;
		// compute the vector Q^t * Y -> B
		for(j = 0; j < rows; j++) 
			sum += colQ[j] * Y[j];
		// back substitution to find the x for Rx = B
		for(j = cols-1; j > i; j--)
			sum -= R[i+j*cols] * B[j];

		B[i] = sum / R[i+i*cols];
	}
}

__global__ void ftest(int diagFlag, int p, int rows, int colsx, int colsy, 
	int rCols, int unrCols, float * obs, int obsDim, 
	float * rCoeffs, int rCoeffsDim, float * unrCoeffs, int unrCoeffsDim, 
	float * rdata, int rdataDim, float * unrdata, int unrdataDim, 
	float * dfStats) // float * dpValues)
{
	int 
		j = blockIdx.x * THREADSPERDIM + threadIdx.x, 
		i = blockIdx.y * THREADSPERDIM + threadIdx.y, 
		idx = i*colsx + j, k, m;
	float 
		kobs, fp = (float) p, frows = (float) rows,
		rSsq, unrSsq,
		rEst, unrEst,
		score = 0.f, 
		* tObs, * tRCoeffs, * tUnrCoeffs, 
		* tRdata, * tUnrdata; 

	if((i >= colsy) || (j >= colsx)) return;
	if((!diagFlag) && (i == j)) {
		dfStats[idx] = 0.f;
		// dpValues[idx] = 0.f;
		return;
	}

	tObs = obs + (i*colsx+j)*obsDim;

	tRCoeffs = rCoeffs + i*rCoeffsDim;
	tRdata = rdata + i*rdataDim;
	
	tUnrCoeffs = unrCoeffs + (i*colsx+j)*unrCoeffsDim;
	tUnrdata = unrdata + (i*colsx+j)*unrdataDim;

	rSsq = unrSsq = 0.f;
	for(k = 0; k < rows; k++) {
		unrEst = rEst = 0.f;
		kobs = tObs[k];
		for(m = 0; m < rCols; m++)
			rEst += tRCoeffs[m] * tRdata[k+m*rows];
		for(m = 0; m < unrCols; m++) 
			unrEst += tUnrCoeffs[m] * tUnrdata[k+m*rows];
		rSsq   += (kobs - rEst) * (kobs - rEst);
		unrSsq += (kobs - unrEst) * (kobs - unrEst);

	}
	score = ((rSsq - unrSsq)*(frows-2.f*fp-1.f)) / (fp*unrSsq);

	if(!isfinite(score))
		score = 0.f;

	dfStats[idx] = score;
/* 	
	float 
		x = score, mfact, alpha, sum = 0.f,
		d1 = (float)p, d2 = (float)rows - 2.f * (float)p - 1.f,
		v = ((float) (d1*x)) / ((float) (d1*x+d2)), 
		a = ((int)d1) / 2, b = ((int)d2) / 2;

	if((int)d1 % 2 != 0) a++;
	if((int)d2 % 2 != 0) b++; 
	if((a < 0) || (b < 0)) return; 

	alpha = a+b-1;
	for(k = a+b-2; (k > 1) && (k > b-2); k--) 
		alpha *= k;

	mfact = a;
	for(m = 2; m < a; m++)
		mfact *= (float)m;

	for(k = -1, m = a; m < a+b; m++, k++) {
		sum += (alpha*__powf(v, (float)m)*__powf(1.f-v, (float)(a+b-1-m))) 
			/ mfact;
		if(b+k <= 0) alpha = 1.f;
		else alpha /= (float)b+k;
		mfact *= m+1;
	}
	dpValues[idx] = 1.f - sum; */
}

void getPValues(int rows, int cols, const float * fStats, int p, int embedRows,
	float * pValues)
{
	float fscore = 0.f;

	for(int i = 0; i < rows; i++) {
		for(int j = 0; j < cols; j++) {
			fscore = fStats[i + j * rows];
			pValues[i + j * rows] = 1.f - (float) pf((double) fscore,
				(double) p, (double)embedRows - 2.0 * (double) p - 1.0,
				1, 0);
		}
	}
}

void granger(int rows, int cols, const float * y, int p, 
	float * fStats, float * pValues)
{

	if(cols < 2) {
		fatal("The Granger test needs at least 2 variables.\n");
		return;
	}
	int
		i, j, k, t = p+1,
		fbytes = sizeof(float),
		embedRows = rows-p, embedCols = t*2;
	float 
		* Y, * rQ, * rR,
		* unrQ, * unrR,
		* restricted, * unrestricted,
		* rdata, * unrdata,
		* dfStats; // * dpValues;
	size_t 
		size = cols*cols*fbytes, partSize = embedRows*size;

	hipMalloc((void **)&Y, embedCols*partSize);

	hipMalloc((void **)&rQ, t*embedRows*cols*fbytes);
	hipMalloc((void **)&rR, t*t*cols*fbytes);
	hipMalloc((void **)&rdata, t*embedRows*cols*fbytes);
	hipMalloc((void **)&unrdata, (embedCols-1)*partSize);
	hipMalloc((void **)&restricted, t*cols*fbytes);
	if( hasCudaError("granger: line 267: gpu memory allocation") ) return;

	int
		Ydim =  embedCols * embedRows,
		rQdim = t * embedRows, rRdim = t * t,
		rdataDim = t*embedRows, restrictedDim = t,
		unrQdim = (embedCols-1) * embedRows, 
		unrRdim = (embedCols-1) * (embedCols-1),
		unrestrictedDim = embedCols-1, unrdataDim = (embedCols-1)*embedRows;
	float 
		* ypos, * rdataPos, * unrdataPos,
		* evenCols;
	int 
		skip = 2*embedRows, colBytes = embedRows*fbytes;
	const float 
		* vectA, * vectB;

	for(i = 0; i < cols; i++) {
		rdataPos = rdata+i*rdataDim;
		evenCols = rdataPos+embedRows;
		vectA = y+i*rows; 
		for(j = 0; j < cols; j++) {
			if(i == j) continue;
			
			ypos = Y+(i*cols+j)*Ydim;
			unrdataPos = unrdata+(i*cols+j)*unrdataDim;

			vectB = y+j*rows;

			for(k = 0; k < p+1; k++) { // produce t subcols
				hipMemcpy(ypos+k*skip, vectA+(p-k), embedRows*fbytes, 
					hipMemcpyHostToDevice);
				hipMemcpy(ypos+k*skip+embedRows, vectB+(p-k), 
					embedRows*fbytes, hipMemcpyHostToDevice);
			}
			hipMemcpy(unrdataPos+embedRows, ypos+skip,
				(embedCols-2)*embedRows*fbytes, hipMemcpyDeviceToDevice);
		}
		// build restricted data from last set of unrestricted data
		// only need one per column, not one for each pairing
		for(k = 0; k < embedCols-2; k+=2) { 
			hipMemcpy(evenCols+(k*embedRows)/2, unrdataPos+(1+k)*embedRows,
				colBytes, hipMemcpyDeviceToDevice);
		}
	}
	if( hasCudaError("granger : mem copy from host to device") ) return;

	int numBlocks;

	numBlocks = cols / THREADSPERDIM;
	if(numBlocks * THREADSPERDIM < cols) numBlocks++;

	dim3 
		dimRGrid(numBlocks), 
		dimRBlock(THREADSPERDIM), 
		dimUnrGrid(numBlocks, numBlocks), 
		dimUnrBlock(THREADSPERDIM, THREADSPERDIM);

	getRestricted<<<dimRGrid, dimRBlock>>>(cols, cols, embedRows, t, rdata, 
		rdataDim, Y, Ydim, rQ, rQdim, rR, rRdim, restricted, restrictedDim);
	if( hasCudaError("granger: getRestricted kernel execution") ) return;

	hipFree(rQ);
	hipFree(rR);

	hipMalloc((void **)&unrQ, (embedCols-1)*partSize);
	hipMalloc((void **)&unrR, (embedCols-1)*(embedCols-1)*size);
	hipMalloc((void **)&unrestricted, (embedCols-1)*size);
	if( hasCudaError("granger: line 336: attemped gpu memory allocation") ) 
		return;

	getUnrestricted<<<dimUnrGrid, dimUnrBlock>>>(cols, cols, embedRows, 
		embedCols-1, 
		unrdata, unrdataDim, Y, Ydim, unrQ, unrQdim, unrR, unrRdim, 
		unrestricted, unrestrictedDim);
	if( hasCudaError("granger : getUnRestricted kernel execution") ) return;

	hipFree(unrQ);
	hipFree(unrR);

	size_t resultSize = cols*cols*fbytes;
	hipMalloc((void **)&dfStats, resultSize);
	// hipMalloc((void **)&dpValues, resultSize);
	if( hasCudaError("granger: line 350: gpu memory allocation") ) return;

	ftest<<<dimUnrGrid, dimUnrBlock>>>(FALSE, p, embedRows, cols, cols, t, 
		embedCols-1, Y, Ydim, restricted, restrictedDim, 
		unrestricted, unrestrictedDim, rdata, rdataDim, unrdata, unrdataDim,
		dfStats); // dpValues); 
	if( hasCudaError("granger : ftest kernel execution") ) return;

	hipMemcpy(fStats, dfStats, resultSize, hipMemcpyDeviceToHost);
	// hipMemcpy(pValues, dpValues, resultSize, hipMemcpyDeviceToHost);
	if( hasCudaError("granger : mem copy device to host") ) return;

	getPValues(cols, cols, fStats, p, embedRows, pValues);

	hipFree(Y);
	hipFree(restricted);
	hipFree(unrestricted);
	hipFree(rdata);
	hipFree(unrdata);
	hipFree(dfStats);
	// hipFree(dpValues);
}

void grangerxy(int rows, int colsx, const float * x, int colsy, 
	const float * y, int p, float * fStats, float * pValues)
{

	if((p < 0) || (rows < 1) || (colsx < 1) || (colsy < 1)) {
		fatal("The Granger XY test needs at least a pair variables.\n");
		return;
	}
	int
		i, j, k, t = p+1,
		fbytes = sizeof(float),
		embedRows = rows-p, embedCols = t*2;
	float 
		* Y, * rQ, * rR,
		* unrQ, * unrR,
		* restricted, * unrestricted,
		* rdata, * unrdata,
		* dfStats; // * dpValues;
	size_t 
		size = colsx*colsy*fbytes, partSize = embedRows*size;

	hipMalloc((void **)&Y, embedCols*partSize);

	hipMalloc((void **)&rQ, t*embedRows*colsy*fbytes);
	hipMalloc((void **)&rR, t*t*colsy*fbytes);
	hipMalloc((void **)&rdata, t*embedRows*colsy*fbytes);
	hipMalloc((void **)&restricted, t*colsy*fbytes);

	hipMalloc((void **)&unrQ, (embedCols-1)*partSize);
	hipMalloc((void **)&unrR, (embedCols-1)*(embedCols-1)*size);
	hipMalloc((void **)&unrestricted, (embedCols-1)*size);
	hipMalloc((void **)&unrdata, (embedCols-1)*partSize);
	checkCudaError("grangerxy : attemped gpu memory allocation");

	int
		Ydim =  embedCols * embedRows,
		rQdim = t * embedRows, rRdim = t * t,
		rdataDim = t*embedRows, restrictedDim = t,
		unrQdim = (embedCols-1) * embedRows, 
		unrRdim = (embedCols-1) * (embedCols-1),
		unrestrictedDim = embedCols-1, unrdataDim = (embedCols-1)*embedRows;
	float 
		* ypos, * rdataPos, * unrdataPos;

	int 
		skip = 2*embedRows, colBytes = embedRows*fbytes;
	const float * vectA, * vectB;
	float * evenCols;

	for(i = 0; i < colsy; i++) {
		rdataPos = rdata+i*rdataDim;
		evenCols = rdataPos+embedRows;
		vectA = y+i*rows; 
		for(j = 0; j < colsx; j++) {
			ypos = Y+(i*colsx+j)*Ydim;
			unrdataPos = unrdata+(i*colsx+j)*unrdataDim;

			vectB = x+j*rows;

			for(k = 0; k < p+1; k++) { // produce t subcols
				hipMemcpy(ypos+k*skip, vectA+(p-k), embedRows*fbytes, 
					hipMemcpyHostToDevice);
				hipMemcpy(ypos+k*skip+embedRows, vectB+(p-k), 
					embedRows*fbytes, hipMemcpyHostToDevice);
			}
			hipMemcpy(unrdataPos+embedRows, ypos+skip,
				(embedCols-2)*embedRows*fbytes, hipMemcpyDeviceToDevice);
		}
		// build restricted data from last set of unrestricted data
		// only need one per column, not one for each pairing
		for(k = 0; k < embedCols-2; k+=2) { 
			hipMemcpy(evenCols+(k*embedRows)/2, unrdataPos+(1+k)*embedRows, 
				colBytes, hipMemcpyDeviceToDevice);
		}
		char errline[16];
		sprintf(errline, "gxy err : %d\n", i);
		if( hasCudaError(errline) ) return;
	}
	checkCudaError("grangerxy : mem copy from host to device");

	int 
		numBlocksX = colsx / THREADSPERDIM,
		numBlocksY = colsy / THREADSPERDIM;

	if(numBlocksX * THREADSPERDIM < colsx) numBlocksX++;
	if(numBlocksY * THREADSPERDIM < colsy) numBlocksY++;

	dim3 
		dimRGrid(numBlocksY), 
		dimRBlock(THREADSPERDIM), 
		dimUnrGrid(numBlocksX, numBlocksY), 
		dimUnrBlock(THREADSPERDIM, THREADSPERDIM);

	getRestricted<<<dimRGrid, dimRBlock>>>(colsx, colsy, embedRows, t, rdata, 
		rdataDim, Y, Ydim, rQ, rQdim, rR, rRdim, restricted, restrictedDim);
	getUnrestricted<<<dimUnrGrid, dimUnrBlock>>>(colsx, colsy, embedRows, 
		embedCols-1, unrdata, unrdataDim, Y, Ydim, unrQ, unrQdim, unrR, 
		unrRdim, unrestricted, unrestrictedDim);
	checkCudaError("grangerxy : kernel execution get(Un)Restricted");

	hipFree(rQ);
	hipFree(unrQ);
	hipFree(rR);
	hipFree(unrR);

	size_t resultSize = colsx*colsy*fbytes;
	hipMalloc((void **)&dfStats, resultSize);
	// hipMalloc((void **)&dpValues, resultSize);
	checkCudaError("grangerxy : attemped gpu memory allocation");

	ftest<<<dimUnrGrid, dimUnrBlock>>>(TRUE, p, embedRows, colsx, colsy, t, 
		embedCols-1, Y, Ydim, restricted, restrictedDim, unrestricted, 
		unrestrictedDim, rdata, rdataDim, unrdata, unrdataDim, 
		dfStats); // dpValues); 
	checkCudaError("grangerxy : kernel execution ftest");

	hipMemcpy(fStats, dfStats, resultSize, hipMemcpyDeviceToHost);
	// hipMemcpy(pValues, dpValues, resultSize, hipMemcpyDeviceToHost);
	checkCudaError("grangerxy : mem copy from device to host");

	getPValues(colsx, colsy, fStats, p, embedRows, pValues);

	hipFree(Y);
	hipFree(restricted);
	hipFree(unrestricted);
	hipFree(rdata);
	hipFree(unrdata);
	hipFree(dfStats);
	// hipFree(dpValues);
}
