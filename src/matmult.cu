#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#include<hip/hip_runtime.h>
#include<hipblas.h>

#include<cuseful.h>

#include<R.h>
#include<Rinternals.h>
#include<R_ext/BLAS.h>

#include<matmult.h>

SEXP gpuMatMult(SEXP a, SEXP b) {
  double
    * xa = REAL(a), * xb = REAL(b),
    * gpua, * gpub, * gpuc;

  SEXP
    dima = getAttrib(a, R_DimSymbol),
    dimb = getAttrib(b, R_DimSymbol);

  int
    rowsa = INTEGER(dima)[0], colsa = INTEGER(dima)[1],
    rowsb = INTEGER(dimb)[0], colsb = INTEGER(dimb)[1];

  hipblasStatus_t stat;
  hipblasHandle_t handle;

  hipError_t cudaStat;

  cudaStat = hipMalloc((void**) &gpua, rowsa * colsa * sizeof(double));
  if (cudaStat != hipSuccess) error("device memory allocation failed");

  cudaStat = hipMalloc((void**) &gpub, rowsb * colsb * sizeof(double));
  if (cudaStat != hipSuccess) error("device memory allocation failed");

  int
    rowsOpA = rowsa, colsOpA = colsa, colsOpB = colsb;

  cudaStat = hipMalloc((void**) &gpuc, rowsOpA * colsOpB * sizeof(double));
  if (cudaStat != hipSuccess) error("device memory allocation failed");

  stat = hipblasCreate(&handle);
  if(stat != HIPBLAS_STATUS_SUCCESS) error("CUBLAS initialization failed\n");

  stat = hipblasSetMatrix(rowsa, colsa, sizeof(double), xa, rowsa,
      gpua, rowsa);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    hipFree(gpuc);
    hipFree(gpub);
    hipFree(gpua);
    hipblasDestroy(handle);
    error("data download failed\n");
  }

  stat = hipblasSetMatrix(rowsb, colsb, sizeof(double), xb, rowsb,
      gpub, rowsb);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    hipFree(gpuc);
    hipFree(gpub);
    hipFree(gpua);
    hipblasDestroy(handle);
    error("data download failed\n");
  }

  const double alpha = 1.0, beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsOpA, colsOpB, colsOpA,
      &alpha, (const double *) gpua, rowsa, (const double *) gpub, rowsb,
      &beta, gpuc, rowsOpA);

  SEXP ab, dimab;
  PROTECT(ab = allocVector(REALSXP, rowsOpA * colsOpB));
  PROTECT(dimab = allocVector(INTSXP, 2));
  INTEGER(dimab)[0] = rowsOpA; INTEGER(dimab)[1] = colsOpB;
  setAttrib(ab, R_DimSymbol, dimab);

  double * xab = REAL(ab);
  stat = hipblasGetMatrix(rowsOpA, colsOpB, sizeof(double), gpuc, rowsOpA,
      xab, rowsOpA);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    hipFree(gpuc);
    hipFree(gpub);
    hipFree(gpua);
    hipblasDestroy(handle);
    error("data upload failed\n");
  }

  hipFree(gpua);
  hipFree(gpub);
  hipFree(gpuc);

  hipblasDestroy(handle);
  UNPROTECT(2);
  return ab;
}
