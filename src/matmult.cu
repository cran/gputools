#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<hipblas.h>
#include<cuseful.h>
#include<matmult.h>

void gpuMatMult(float * a, int rowsa, int colsa, 
	float * b, int rowsb, int colsb, float * c)
{
	if(colsa != rowsb) {
		fprintf(stderr, "error: matrix dimensions mismatched for matrix multiplication\n"); 
		return;
	}

	float
		* gpua, * gpub, * gpuc;

	cublasInit();
	checkCublasError("gpuMatMult device initialization");

	cublasAlloc(rowsa*colsa, sizeof(float), (void **) &gpua);
	cublasAlloc(rowsb*colsb, sizeof(float), (void **) &gpub);
	cublasAlloc(rowsa*colsb, sizeof(float), (void **) &gpuc);
	checkCublasError("gpuMatMult memory allocation");

	hipblasSetVector(rowsa*colsa, sizeof(float), a, 1, gpua, 1);
	hipblasSetVector(colsa*colsb, sizeof(float), b, 1, gpub, 1);
	checkCublasError("gpuMatMult write to gpu memory");

	hipblasSgemm('N', 'N', rowsa, colsb, colsa, 1.0, gpua, rowsa, gpub, colsa, 
		0.0, gpuc, rowsa);
	checkCublasError("gpuMatMult gpu routine execution");

	hipblasGetVector(rowsa*colsb, sizeof(float), gpuc, 1, c, 1);
	checkCublasError("gpuMatMult read from gpu memory");

	cublasFree(gpua);
	cublasFree(gpub);
	cublasFree(gpuc);
	cublasShutdown();
}

void gpu16MatMult(float * a, int rowsa, int colsa, 
	float * b, int rowsb, int colsb, float * c)
{
	if(colsa != rowsb) {
		fprintf(stderr, "error: matrix dimensions mismatched for matrix multiplication\n"); 
		return;
	}
	float
		* bigA, * bigB, * bigC,
		* gpua, * gpub, * gpuc;
	int
		bigRowsA, bigColsA,	// we are gonna blow these up to
		bigRowsB, bigColsB; // the nearest power of 16 to help cuda

	bigRowsA = ((rowsa >> 4) + ((rowsa & 15)? 1:0)) << 4;
	bigColsA = ((colsa >> 4) + ((colsa & 15)? 1:0)) << 4;
	bigRowsB = ((rowsb >> 4) + ((rowsb & 15)? 1:0)) << 4;
	bigColsB = ((colsb >> 4) + ((colsb & 15)? 1:0)) << 4;

	cublasInit();
	checkCublasError("gpu16MatMult device initialization");

	cublasAlloc(bigRowsA*bigColsA, sizeof(float), (void **) &gpua);
	cublasAlloc(bigRowsB*bigColsB, sizeof(float), (void **) &gpub);
	cublasAlloc(bigRowsA*bigColsB, sizeof(float), (void **) &gpuc);
	checkCublasError("gpu16MatMult memory allocation");

	bigA = (float *) calloc(bigRowsA*bigColsA, sizeof(float));
	for(int i = 0; i < colsa; i++)
		memcpy(&bigA[i*bigRowsA], &a[i*rowsa], rowsa*sizeof(float));

	bigB = (float *) calloc(bigRowsB*bigColsB, sizeof(float));
	for(int i = 0; i < colsb; i++)
		memcpy(&bigB[i*bigRowsB], &b[i*rowsb], rowsb*sizeof(float));

	hipblasSetVector(bigRowsA*bigColsA, sizeof(float), bigA, 1, gpua, 1);
	hipblasSetVector(bigRowsB*bigColsB, sizeof(float), bigB, 1, gpub, 1);
	checkCublasError("gpu16MatMult write to gpu memory");

	hipblasSgemm('N', 'N', bigRowsA, bigColsB, bigColsA, 1.0, gpua, bigRowsA,
		gpub, bigRowsB, 0.0, gpuc, bigRowsA);
	checkCublasError("gpu16MatMult gpu routine execution");

	bigC = (float *) calloc(bigRowsA*bigColsB, sizeof(float));
	hipblasGetVector(bigRowsA*bigColsB, sizeof(float), gpuc, 1, bigC, 1);
	checkCublasError("gpu16MatMult read from gpu memory");

	for(int i = 0; i < colsb; i++)
		memcpy(&c[i*rowsa], &bigC[i*bigRowsA], rowsa*sizeof(float));

	cublasFree(gpua);
	cublasFree(gpub);
	cublasFree(gpuc);
	cublasShutdown();
	free(bigA);
	free(bigB);
	free(bigC);
}
