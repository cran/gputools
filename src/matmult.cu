#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#include<hip/hip_runtime.h>
#include<hipblas.h>

#include<cuseful.h>
#include<matmult.h>

RcppExport SEXP gpuMatMult(SEXP a, SEXP b) {
	Rcpp::NumericMatrix xa(a);
	Rcpp::NumericMatrix xb(b);

	double
		* gpua, * gpub, * gpuc;

	int
		rowsa = xa.nrow(), colsa = xa.ncol(),
		rowsb = xb.nrow(), colsb = xb.ncol();

	hipblasStatus_t stat;
	hipblasHandle_t handle;

	hipError_t cudaStat;

	cudaStat = hipMalloc((void**) &gpua, rowsa * colsa * sizeof(double));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return NULL;
	}  

	cudaStat = hipMalloc((void**) &gpub, rowsb * colsb * sizeof(double));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return NULL;
	}  

	hipblasOperation_t opA = tpA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	hipblasOperation_t opB = tpB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

	int rowsOpA = tpA ? colsa : rowsa;
	int colsOpA = tpA ? rowsa : colsa;
	int colsOpB = tpB ? rowsb : colsb;

	cudaStat = hipMalloc((void**) &gpuc, rowsOpA * colsOpB * sizeof(double));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return NULL;
	}  

	stat = hipblasCreate(&handle);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization failed\n");
		return NULL;
	}

	stat = hipblasSetMatrix(rowsa, colsa, sizeof(double), xa, rowsa,
		gpua, rowsa);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data download failed\n");
		hipFree(gpuc);
		hipFree(gpub);
		hipFree(gpua);
		hipblasDestroy(handle);
		return NULL;
	}

	Rcpp::NumericMatrix xb(b);
	stat = hipblasSetMatrix(rowsb, colsb, sizeof(double), xb, rowsb,
		gpub, rowsb);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data download failed\n");
		hipFree(gpuc);
		hipFree(gpub);
		hipFree(gpua);
		hipblasDestroy(handle);
		return NULL;
	}

	const double alpha = 1.0, beta = 0.0;
	hipblasDgemm(handle, opA, opB, rowsOpA, colsOpB, colsOpA, &alpha,
		(const double *) gpua, rowsa, (const double *) gpub, rowsb,
		&beta, gpuc, rowsOpA);

	Rcpp::NumericMatrix xab(rowsa, colsb);
	stat = hipblasGetMatrix(rowsOpA, colsOpB, sizeof(double), gpuc, rowsOpA,
		xab, rowsOpA);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data upload failed\n");
		hipFree(gpuc);
		hipFree(gpub);
		hipFree(gpua);
		hipblasDestroy(handle);
		return NULL;
	}

	hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	hipblasDestroy(handle);
	return xab;
}
