#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hipblas.h>
#include<cuseful.h>
#include<R.h>
#include<correlation.h>

#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuMeans(const float * vectsA, size_t na, 
	const float * vectsB, size_t nb, size_t dim, 
	float * means, float * numPairs)
{
	size_t 
		offset, stride,
		bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
	float a, b;

	__shared__ float 
		threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS],
		count[NUMTHREADS];

	if((bx >= na) || (by >= nb))
		return;

	threadSumsA[tx] = 0.f;
	threadSumsB[tx] = 0.f;
	count[tx] = 0.f;

	for(offset = tx; offset < dim; offset += NUMTHREADS) {
		a = vectsA[bx * dim + offset];
		b = vectsB[by * dim + offset];
		if(!(isnan(a) || isnan(b))) {
			threadSumsA[tx] += a;
			threadSumsB[tx] += b;
			count[tx] += 1.f;
		}
	}
	__syncthreads();
    
	for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
		if(tx < stride) {
			threadSumsA[tx] += threadSumsA[tx + stride];
			threadSumsB[tx] += threadSumsB[tx + stride];
			count[tx] += count[tx+stride];
		}
		__syncthreads();
	}
	if(tx == 0) {
		means[bx*nb*2+by*2] = threadSumsA[0] / count[0];
		means[bx*nb*2+by*2+1] = threadSumsB[0] / count[0];
		numPairs[bx*nb+by] = count[0];
	}
}

__global__ void gpuSD(const float * vectsA, size_t na,
	const float * vectsB, size_t nb, size_t dim, 
	const float * means, const float * numPairs, float * sds)
{
	size_t 
		offset, stride,
		tx = threadIdx.x, 
		bx = blockIdx.x, by = blockIdx.y;
	float 
		a, b,
		termA, termB;
	__shared__ float 
		meanA, meanB, n,
		threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];

	if((bx >= na) || (by >= nb))
		return;

	if(tx == 0) {
		meanA = means[bx*nb*2+by*2];	
		meanB = means[bx*nb*2+by*2+1];	
		n = numPairs[bx*nb+by]; 
	}
	__syncthreads();

	threadSumsA[tx] = 0.f;
	threadSumsB[tx] = 0.f;
	for(offset = tx; offset < dim; offset += NUMTHREADS) {
		a = vectsA[bx * dim + offset];
		b = vectsB[by * dim + offset];
		if(!(isnan(a) || isnan(b))) {
			termA = a - meanA;
			termB = b - meanB;
			threadSumsA[tx] += termA * termA;
			threadSumsB[tx] += termB * termB;
		}
	}
	__syncthreads();

	for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
		if(tx < stride) {
			threadSumsA[tx] += threadSumsA[tx + stride];
			threadSumsB[tx] += threadSumsB[tx + stride];
		}
		__syncthreads();
	}
	if(tx == 0) {
		sds[bx*nb*2+by*2]   = sqrtf(threadSumsA[0] / (n - 1.f));
		sds[bx*nb*2+by*2+1] = sqrtf(threadSumsB[0] / (n - 1.f));
	}
}

__global__ void gpuPMCC(const float * vectsa, size_t na,
	const float * vectsb, size_t nb, size_t dim,
	const float * numPairs, const float * means, const float * sds,
	float * correlations) 
{
	size_t 
		offset, stride,
		x = blockIdx.x, y = blockIdx.y, 
		tx = threadIdx.x;
	float 
		a, b, n, scoreA, scoreB;
    __shared__ float 
		meanA, meanB, 
		sdA, sdB, 
		threadSums[NUMTHREADS];

	if((x >= na) || (y >= nb))
		return;

	if(tx == 0) {
		meanA = means[x*nb*2+y*2];
		meanB = means[x*nb*2+y*2+1];	
		sdA = sds[x*nb*2+y*2];
		sdB = sds[x*nb*2+y*2+1];	
		n = numPairs[x*nb+y]; 
	}
	__syncthreads();

	threadSums[tx] = 0.f;
	for(offset = tx; offset < dim; offset += NUMTHREADS) {
		a = vectsa[x * dim + offset];
		b = vectsb[y * dim + offset];
		if(!(isnan(a) || isnan(b))) {
			scoreA = (a - meanA) / sdA; 
			scoreB = (b - meanB) / sdB;
			threadSums[tx] += scoreA * scoreB;
		}
	}
	__syncthreads();

	for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
		if(tx < stride) threadSums[tx] += threadSums[tx + stride];
		__syncthreads();
	}
	if(tx == 0) correlations[x*nb+y] = threadSums[0] / (n - 1.f);
}

__global__ void gpuMeansNoTest(const float * vectsA, size_t na, 
	const float * vectsB, size_t nb, size_t dim, 
	float * means, float * numPairs)
{
	size_t 
		offset, stride,
		bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
	float a, b;

	__shared__ float 
		threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS],
		count[NUMTHREADS];

	if((bx >= na) || (by >= nb))
		return;

	threadSumsA[tx] = 0.f;
	threadSumsB[tx] = 0.f;
	count[tx] = 0.f;

	for(offset = tx; offset < dim; offset += NUMTHREADS) {
		a = vectsA[bx * dim + offset];
		b = vectsB[by * dim + offset];

		threadSumsA[tx] += a;
		threadSumsB[tx] += b;
		count[tx] += 1.f;
	}
	__syncthreads();
    
	for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
		if(tx < stride) {
			threadSumsA[tx] += threadSumsA[tx + stride];
			threadSumsB[tx] += threadSumsB[tx + stride];
			count[tx] += count[tx+stride];
		}
		__syncthreads();
	}
	if(tx == 0) {
		means[bx*nb*2+by*2] = threadSumsA[0] / count[0];
		means[bx*nb*2+by*2+1] = threadSumsB[0] / count[0];
		numPairs[bx*nb+by] = count[0];
	}
}

__global__ void gpuSDNoTest(const float * vectsA, size_t na,
	const float * vectsB, size_t nb, size_t dim, 
	const float * means, const float * numPairs, float * sds)
{
	size_t 
		offset, stride,
		tx = threadIdx.x, 
		bx = blockIdx.x, by = blockIdx.y;
	float 
		a, b,
		termA, termB;
	__shared__ float 
		meanA, meanB, n,
		threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];

	if((bx >= na) || (by >= nb))
		return;

	if(tx == 0) {
		meanA = means[bx*nb*2+by*2];	
		meanB = means[bx*nb*2+by*2+1];	
		n = numPairs[bx*nb+by]; 
	}
	__syncthreads();

	threadSumsA[tx] = 0.f;
	threadSumsB[tx] = 0.f;
	for(offset = tx; offset < dim; offset += NUMTHREADS) {
		a = vectsA[bx * dim + offset];
		b = vectsB[by * dim + offset];

		termA = a - meanA;
		termB = b - meanB;
		threadSumsA[tx] += termA * termA;
		threadSumsB[tx] += termB * termB;
	}
	__syncthreads();

	for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
		if(tx < stride) {
			threadSumsA[tx] += threadSumsA[tx + stride];
			threadSumsB[tx] += threadSumsB[tx + stride];
		}
		__syncthreads();
	}
	if(tx == 0) {
		sds[bx*nb*2+by*2]   = sqrtf(threadSumsA[0] / (n - 1.f));
		sds[bx*nb*2+by*2+1] = sqrtf(threadSumsB[0] / (n - 1.f));
	}
}

__global__ void gpuPMCCNoTest(const float * vectsa, size_t na,
	const float * vectsb, size_t nb, size_t dim,
	const float * numPairs, const float * means, const float * sds,
	float * correlations) 
{
	size_t 
		offset, stride,
		x = blockIdx.x, y = blockIdx.y, 
		tx = threadIdx.x;
	float 
		a, b, n, scoreA, scoreB;
    __shared__ float 
		meanA, meanB, 
		sdA, sdB, 
		threadSums[NUMTHREADS];

	if((x >= na) || (y >= nb))
		return;

	if(tx == 0) {
		meanA = means[x*nb*2+y*2];
		meanB = means[x*nb*2+y*2+1];	
		sdA = sds[x*nb*2+y*2];
		sdB = sds[x*nb*2+y*2+1];	
		n = numPairs[x*nb+y]; 
	}
	__syncthreads();

	threadSums[tx] = 0.f;
	for(offset = tx; offset < dim; offset += NUMTHREADS) {
		a = vectsa[x * dim + offset];
		b = vectsb[y * dim + offset];
		
		scoreA = (a - meanA) / sdA; 
		scoreB = (b - meanB) / sdB;
		threadSums[tx] += scoreA * scoreB;
	}
	__syncthreads();

	for(stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
		if(tx < stride) threadSums[tx] += threadSums[tx + stride];
		__syncthreads();
	}
	if(tx == 0) correlations[x*nb+y] = threadSums[0] / (n - 1.f);
}

__global__ void gpuSignif(const float * gpuNumPairs, 
	const float * gpuCorrelations, size_t n, float * gpuTScores)
{
	size_t 
		i, start,
		bx = blockIdx.x, tx = threadIdx.x;
	float 
		radicand, cor, npairs;

	start = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;
	for(i = 0; i < THREADWORK; i++) {
		if(start+i >= n)
			break;

		npairs = gpuNumPairs[start+i];
		cor = gpuCorrelations[start+i];
		radicand = (npairs - 2.f) / (1.f - cor * cor);
		gpuTScores[start+i] = cor * sqrtf(radicand);
	}
}

__host__ void testSignif(const float * goodPairs, const float * coeffs, 
	size_t n, float * tscores)
{
	size_t
		fbytes = sizeof(float), size = n*fbytes;
	float
		* gpuPairs, * gpuCoeffs, * gpuTs;


	hipMalloc((void **)&gpuPairs, size);
	hipMalloc((void**)&gpuCoeffs, size);
	hipMalloc((void**)&gpuTs, size);

	hipMemcpy(gpuPairs, goodPairs, size, hipMemcpyHostToDevice);
	hipMemcpy(gpuCoeffs, coeffs, size, hipMemcpyHostToDevice);

	size_t nblocks = n / (NUMTHREADS*THREADWORK);
	if(nblocks*(NUMTHREADS*THREADWORK) < n) nblocks++;
	dim3 
		tblock(NUMTHREADS), tgrid(nblocks);

	gpuSignif<<<tgrid, tblock>>>(gpuPairs, gpuCoeffs, n, gpuTs);

	hipMemcpy(tscores, gpuTs, size, hipMemcpyDeviceToHost);
	hipFree(gpuPairs);
	hipFree(gpuCoeffs);
	hipFree(gpuTs);
}

void hostSignif(const float * goodPairs, const float * coeffs, size_t n, 
	float * tscores)
{
	float cor, radicand;
	for(size_t i = 0; i < n; i++) {
		cor = coeffs[i];
		if(cor >= 0.999f) 
			tscores[i] = 10000.0f;
		else {
			radicand = (goodPairs[i] - 2.f) / (1.f - cor * cor);
			tscores[i] = cor * sqrtf(radicand);
		}
	}
}

__host__ void pmcc(UseObs whichObs, const float * vectsa, size_t na,
	const float * vectsb, size_t nb, size_t dim, float * numPairs, 
	float * correlations, float * signifs)
{
	size_t 
		fbytes = sizeof(float);
	int 
		same = (vectsa == vectsb);
	float
		* gpuVA, * gpuVB,
		* gpuNumPairs, * gpumean, * gpuSds, 
		* gpuCorrelations;
	dim3 
		block(NUMTHREADS), grid(na, nb);

	hipMalloc((void **)&gpuNumPairs, na*nb*fbytes);
	hipMalloc((void **)&gpumean, na*nb*2*fbytes);
	hipMalloc((void **)&gpuSds, na*nb*2*fbytes);
	hipMalloc((void**)&gpuCorrelations, na*nb*fbytes);

	hipMalloc((void**)&gpuVA, na*dim*fbytes);
	hipMemcpy(gpuVA, vectsa, na*dim*fbytes, hipMemcpyHostToDevice);

	if(!same) { 
		hipMalloc((void**)&gpuVB, nb*dim*fbytes);
		hipMemcpy(gpuVB, vectsb, nb*dim*fbytes, hipMemcpyHostToDevice);
	} else {
		gpuVB = gpuVA;
	}
	checkCudaError("PMCC function : malloc and memcpy");

	switch(whichObs) {
		case pairwiseComplete:
			gpuMeans<<<grid, block>>>(gpuVA, na, gpuVB, nb, dim, gpumean,
				gpuNumPairs);
			hipDeviceSynchronize();
			gpuSD<<<grid, block>>>(gpuVA, na, gpuVB, nb, dim, gpumean, gpuNumPairs, 
				gpuSds);
			hipDeviceSynchronize();
			gpuPMCC<<<grid, block>>>(gpuVA, na, gpuVB, nb, dim, gpuNumPairs,
				gpumean, gpuSds, gpuCorrelations); 
			break;
		default:
			gpuMeansNoTest<<<grid, block>>>(gpuVA, na, gpuVB, nb, dim, gpumean,
				gpuNumPairs);
			hipDeviceSynchronize();
			gpuSDNoTest<<<grid, block>>>(gpuVA, na, gpuVB, nb, dim, gpumean,
				gpuNumPairs, gpuSds);
			hipDeviceSynchronize();
			gpuPMCCNoTest<<<grid, block>>>(gpuVA, na, gpuVB, nb, dim, gpuNumPairs,
				gpumean, gpuSds, gpuCorrelations); 
	}

	hipMemcpy(correlations, gpuCorrelations, na*nb*fbytes, 
		hipMemcpyDeviceToHost);
	hipMemcpy(numPairs, gpuNumPairs, na*nb*fbytes, hipMemcpyDeviceToHost);
	checkCudaError("PMCC function : kernel finish and memcpy");

	hostSignif(numPairs, correlations, na*nb, signifs);
    
	// Free allocated space
	hipFree(gpuNumPairs);
	hipFree(gpumean);
	hipFree(gpuSds);
	hipFree(gpuCorrelations);
	hipFree(gpuVA);
	if(!same) {
		hipFree(gpuVB);
	}
}

void setDevice(int device) {
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if((device < 0) || (device >= deviceCount))
		fatal("The gpu id number is not valid.");

	hipSetDevice(device);
	checkCudaError("setDevice function : choosing gpu");
}

void getDevice(int * device) {
	hipGetDevice(device);
}

void findMinMax(const int * data, size_t n, int * minVal, int * maxVal) {
	size_t i;
	*minVal = *maxVal = data[0];
	for(i = 1; i < n; i++) {
		if(data[i] < *minVal) *minVal = data[i];
		if(data[i] > *maxVal) *maxVal = data[i];
	}
}

void getData(const int * images, 
	const int * xcoords, const int * ycoords, const int * zcoords,
	const int * mins, const int * maxes,
	const float * evs, size_t numrows, size_t numimages, float * output)
{
	int 
		xmin, ymin, zmin,
		xmax, ymax, zmax,
		prevImg;
	size_t
		i, j = 0,
		nx, ny, nz,
		x, y, z;

	xmin = mins[0];
	ymin = mins[1];
	zmin = mins[2];	

	xmax = maxes[0];
	ymax = maxes[1];
	zmax = maxes[2];	

	nx = 1+abs(xmax - xmin);
	ny = 1+abs(ymax - ymin);
	nz = 1+abs(zmax - zmin);

	prevImg = images[0];
	for(i = 0; (i < numrows) && (j < numimages); i++) {
		if(prevImg != images[i]) {
			prevImg = images[i];
			j++;
		}
		x = xcoords[i] - xmin;
		y = ycoords[i] - ymin;
		z = zcoords[i] - zmin;
		output[j*nx*ny*nz+x*ny*nz+y*nz+z] = evs[i];
	}
}

size_t parseResults(const int * imageList1, size_t numImages1, 
	const int * imageList2, size_t numImages2,
	int structureid,
	double cutCorrelation, int cutPairs,
	const double * correlations, const double * signifs, const int * numPairs, 
	double * results)
{
	size_t 
		i, j, pos, nrows = 0,
		readpos, writepos;
	double 
		sid = (double) structureid, 
		npair, img1, img2, coeff, signif;

	for(i = 0; i < numImages1; i++) {
		pos = i*numImages2;
		img1 = (double) imageList1[i];
		for(j = 0; j < numImages2; j++) {
			readpos = pos + j;

			npair = (double) numPairs[readpos];
			coeff = (double) correlations[readpos];
			signif = (double) signifs[readpos];
			img2 = (double) imageList2[j];

			if((fabs(coeff) >= cutCorrelation) && (npair >= cutPairs) 
				&& (img1 < img2) && isSignificant(signif, numPairs[readpos]-2))
			{
				writepos = 6*nrows;
				results[writepos]   = img1;
				results[writepos+1] = img2;
				results[writepos+2] = sid;
				results[writepos+3] = coeff;
				results[writepos+4] = signif;
				results[writepos+5] = npair;
				nrows++;
			}
		}
	}
	return nrows;
}

int isSignificant(double signif, int df) {
	double tcutoffs[49] = {
		// cuttoffs for degrees of freedom <= 30
		637.000, 31.600, 2.920, 8.610, 6.869, 5.959, 5.408, 5.041, 4.781, 
		4.587, 4.437, 4.318, 4.221, 4.140, 4.073, 4.015, 3.965, 3.922, 
		3.883, 3.850, 3.819, 3.792, 3.768, 3.745, 3.725, 3.707, 3.690, 
		3.674, 3.659, 3.646,
		// cuttoffs for even degrees of freedom > 30 but <= 50
		3.622, 3.601, 3.582, 3.566, 3.551, 3.538, 3.526, 3.515, 3.505, 3.496, 
		// 55 <= df <= 70 by 5s
		3.476, 3.460, 3.447, 3.435, 
		3.416, // 80
		3.390, // 100
		3.357, // 150
		3.340, // 200
		3.290  // > 200
	};

	size_t index = 0;
	if(df <= 0) return 0;
	else if(df <= 30) index = df - 1;
	else if(df <= 50) index = 30 + (df + (df%2) - 32) / 2;
	else if(df <= 70) {
		if(df <= 55) index = 40;
		else if(df <= 60) index = 41;
		else if(df <= 65) index = 42;
		else if(df <= 70) index = 43;
	}
	else if(df <= 80) index = 44;
	else if(df <= 100) index = 45;
	else if(df <= 150) index = 46;
	else if(df <= 200) index = 47;
	else if(df > 200) index = 48;

	if(fabs(signif) < tcutoffs[index]) return FALSE;

	return TRUE;
}

__device__ int dIsSignificant(float signif, int df) {
	float tcutoffs[49] = {
		// cuttoffs for degrees of freedom <= 30
		637.000, 31.600, 2.920, 8.610, 6.869, 5.959, 5.408, 5.041, 4.781, 
		4.587, 4.437, 4.318, 4.221, 4.140, 4.073, 4.015, 3.965, 3.922, 
		3.883, 3.850, 3.819, 3.792, 3.768, 3.745, 3.725, 3.707, 3.690, 
		3.674, 3.659, 3.646,
		// cuttoffs for even degrees of freedom > 30 but <= 50
		3.622, 3.601, 3.582, 3.566, 3.551, 3.538, 3.526, 3.515, 3.505, 3.496, 
		// 55 <= df <= 70 by 5s
		3.476, 3.460, 3.447, 3.435, 
		3.416, // 80
		3.390, // 100
		3.357, // 150
		3.340, // 200
		3.290  // > 200
	};

	size_t index = 0;
	if(df <= 0) return 0;
	else if(df <= 30) index = df - 1;
	else if(df <= 50) index = 30 + (df + (df%2) - 32) / 2;
	else if(df <= 70) {
		if(df <= 55) index = 40;
		else if(df <= 60) index = 41;
		else if(df <= 65) index = 42;
		else if(df <= 70) index = 43;
	}
	else if(df <= 80) index = 44;
	else if(df <= 100) index = 45;
	else if(df <= 150) index = 46;
	else if(df <= 200) index = 47;
	else if(df > 200) index = 48;

	if(fabsf(signif) < tcutoffs[index]) return FALSE;

	return TRUE;
}

size_t signifFilter(const double * data, size_t rows, double * results)
{
	size_t i, inrow, outrow, rowcount = 0;
	double tscore, radicand, cor, npairs;

	if(results == NULL) {
		fprintf(stderr, "signifFilter : no ram set aside for results\n");
		exit(EXIT_FAILURE);
	}

	for(i = 0; i < rows; i++) {
		inrow = i * 5;
		outrow = rowcount * 6;	

		cor = data[inrow+3];
		npairs = data[inrow+4];

		if(cor >= 0.999) 
			tscore = 10000.0;
		else {
			radicand = (npairs - 2.0) / (1.0 - cor * cor);
			tscore = cor * sqrt(radicand);
		}
		if(isSignificant(tscore, (int)(npairs-2))) {
			results[outrow]   = data[inrow];
			results[outrow+1] = data[inrow+1];
			results[outrow+2] = data[inrow+2];
			results[outrow+3] = cor;
			results[outrow+4] = tscore;
			results[outrow+5] = npairs;
			rowcount++;
		}
	}
	return rowcount;
}

__global__ void dUpdateSignif(const float * gpuData, size_t n, 
	float * gpuResults)
{
	size_t 
		i, start, inrow, outrow, 
		bx = blockIdx.x, tx = threadIdx.x;
	float 
		radicand, cor, npairs, tscore;

	start = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;
	
	for(i = 0; i < THREADWORK; i++) {
		if(start+i > n) break;

		inrow = (start+i)*5;
		outrow = (start+i)*6;

		cor = gpuData[inrow+3];
		npairs = gpuData[inrow+4];

		if(cor >= 0.999) 
			tscore = 10000.0;
		else {
			radicand = (npairs - 2.f) / (1.f - cor * cor);
			tscore = cor * sqrtf(radicand);
		}
		if(dIsSignificant(tscore, (int)npairs)) {
			gpuResults[outrow]   = gpuData[inrow];
			gpuResults[outrow+1] = gpuData[inrow+1];
			gpuResults[outrow+2] = gpuData[inrow+2];
			gpuResults[outrow+3] = cor;
			gpuResults[outrow+4] = tscore;
			gpuResults[outrow+5] = npairs;
		} else {
			gpuResults[outrow] = -1.f;
		}
	}
}

__host__ void updateSignifs(const float * data, size_t n, float * results)
{
	size_t
		fbytes = sizeof(float), size = n*fbytes,
		insize = 5*size, outsize = 6*size;
	float
		* gpuData, * gpuResults;

	hipMalloc((void **)&gpuData, insize);
	hipMalloc((void**)&gpuResults, outsize);
	checkCudaError("updateSignifs function : gpu out of RAM");

	hipMemcpy(gpuData, data, insize, hipMemcpyHostToDevice);

	size_t nblocks = n / (NUMTHREADS*THREADWORK);
	if(nblocks*(NUMTHREADS*THREADWORK) < n) nblocks++;
	dim3 
		tblock(NUMTHREADS), tgrid(nblocks);

	dUpdateSignif<<<tgrid, tblock>>>(gpuData, n, gpuResults);
	checkCudaError("updateSignifs function : trouble executing kernel on gpu");

	hipMemcpy(results, gpuResults, outsize, hipMemcpyDeviceToHost);
	hipFree(gpuData);
	hipFree(gpuResults);
	checkCudaError("updateSignifs function : trouble reading from gpu");
}

size_t gpuSignifFilter(const float * data, size_t rows, float * results)
{
	size_t 
		i, rowbytes = 6*sizeof(float),
		inrow, outrow, rowcount = 0;

	if(results == NULL) {
		fprintf(stderr, "signifFilter : no ram set aside for results\n");
		exit(EXIT_FAILURE);
	}

	updateSignifs(data, rows, results);

	for(i = 0; i < rows; i++) {
		inrow = i*6;
		outrow = rowcount*6;	

		if(results[inrow] == -1.f) continue;

		memcpy(results+outrow, results+inrow, rowbytes); 
		rowcount++;
	}
	return rowcount;
}

__global__ void noNAsPmccMeans(int nRows, int nCols, float * a, float * means)
{
	int
		col = blockDim.x * blockIdx.x + threadIdx.x,
		inOffset = col * nRows,
		outOffset = threadIdx.x * blockDim.y,
		j = outOffset + threadIdx.y;
	float sum = 0.f;

	if(col >= nCols) return;

	__shared__ float threadSums[NUMTHREADS*NUMTHREADS];

	for(int i = threadIdx.y; i < nRows; i += blockDim.y)
		sum += a[inOffset + i];

	threadSums[j] = sum;
	__syncthreads();

	for(int i = blockDim.y >> 1; i > 0; i >>= 1) {
		if(threadIdx.y < i) {
			threadSums[outOffset+threadIdx.y] 
				+= threadSums[outOffset+threadIdx.y + i];
		}
		__syncthreads();
	}
	if(threadIdx.y == 0)
		means[col] = threadSums[outOffset] / (float)nRows;
}

void cublasPMCC(const float * sampsa, size_t numSampsA, const float * sampsb, 
	size_t numSampsB, size_t sampSize, float * res)
{
	int 
		same = (sampsa == sampsb);
	float
		* gpua, * gpub, * gpuRes,
		* aRecipSD = Calloc(numSampsA, float);

	cublasInit();
	cublasAlloc(numSampsA*sampSize, sizeof(float), (void **)&gpua);

	checkCublasError("PMCC : alloc for input A");
	hipblasSetVector(numSampsA*sampSize, sizeof(float), sampsa, 1, gpua, 1);

	float * gpuaRecipSD;
	cublasAlloc(numSampsA, sizeof(float), (void **)&gpuaRecipSD);

	dim3
		dimBlock(NUMTHREADS, NUMTHREADS);

	int numBlocks = numSampsA / NUMTHREADS;
	if(numBlocks * NUMTHREADS < numSampsA)
		numBlocks++;

	checkCublasError("PMCC : alloc and set workspace for A");

	for(size_t i = 0; i < sampSize; i++) // subtract mean from each sample
		hipblasSaxpy(numSampsA, -1.f, gpuaRecipSD, 1, gpua+i, sampSize);
	cublasFree(gpuaRecipSD);

	for(size_t i = 0; i < numSampsA; i++) { // sum of squares
		float denom = 
			hipblasSdot(sampSize, gpua+i*sampSize, 1, gpua+i*sampSize, 1);
		aRecipSD[i] = 1.f / sqrtf(denom);
	}
	for(size_t i = 0; i < numSampsA; i++) // div each sample by sqrt of s-o-s
		hipblasSscal(sampSize, aRecipSD[i], gpua+i*sampSize, 1); 
	Free(aRecipSD);
	checkCublasError("PMCC : vector ops for A");

	if(!same) {
		float * bRecipSD = Calloc(numSampsB, float);
	
		cublasAlloc(numSampsB*sampSize, sizeof(float), (void **)&gpub);
		hipblasSetVector(numSampsB*sampSize, sizeof(float), sampsb, 1, gpub, 1);

		float * gpubRecipSD;
		cublasAlloc(numSampsB, sizeof(float), (void **)&gpubRecipSD);

		dim3 dimBlock(NUMTHREADS, NUMTHREADS);
		int numBlocks = numSampsB / NUMTHREADS;
		if(numBlocks * NUMTHREADS < numSampsB)
			numBlocks++;

		noNAsPmccMeans<<<numBlocks, dimBlock>>>(sampSize, numSampsB, gpub, 
			gpubRecipSD);
	
		for(size_t i = 0; i < sampSize; i++) // subtract mean
			hipblasSaxpy(numSampsB, -1.f, gpubRecipSD, 1, gpub+i, sampSize);
		cublasFree(gpubRecipSD);
	
		for(size_t i = 0; i < numSampsB; i++) { // sum of squares
			float denom = 
				hipblasSdot(sampSize, gpub+i*sampSize, 1, 
					gpub+i*sampSize, 1);
			bRecipSD[i] = 1.f / sqrtf(denom);
		}
		for(size_t i = 0; i < numSampsB; i++) // div by s-o-s
			hipblasSscal(sampSize, bRecipSD[i], gpub+i*sampSize, 1); 
		Free(bRecipSD);
		checkCublasError("PMCC : setup for matrix B");
	} else {
		gpub = gpua;
	}

	cublasAlloc(numSampsA*numSampsB, sizeof(float), (void **)&gpuRes);
	hipblasSgemm('T', 'N', numSampsB, numSampsA, sampSize, 1.f, 
		gpub, sampSize, gpua, sampSize, 0.f, gpuRes, numSampsB); 
		// each entry : sum of prod of standard scores
	hipblasGetVector(numSampsA*numSampsB, sizeof(float), gpuRes, 1, res, 1);
	checkCublasError("PMCC : alloc, matrix mult and get result");
	cublasFree(gpuRes);
	cublasShutdown();
}

double hostKendall(const float * X, const float * Y, size_t n) {
	float concordant, discordant, denom;

	concordant = discordant = 0.0;
	for(size_t i = 0; i < n; i++) {
		for(size_t j = i+1; j < n; j++) {
			if((X[j] > X[i]) && (Y[j] > Y[i])) 
				concordant = concordant + 1.0;
			else if((X[j] < X[i]) && (Y[j] < Y[i])) 
				concordant = concordant + 1.0;
		}
	}
	denom = (double) n;
	denom = denom*(denom - 1.0) / 2.0;
	discordant = denom - concordant;
	return (double)((double)(concordant - discordant)) / ((double)denom);
}

void permHostKendall(const float * a, size_t na, const float * b, size_t nb,
	size_t sampleSize, double * results)
{
	for(size_t i = 0; i < nb; i++) {
		for(size_t j = 0; j < na; j++) {
			results[i*na+j] = hostKendall(a+j*sampleSize, b+i*sampleSize, 
				sampleSize);
		}
	}
}
